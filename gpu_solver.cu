#include "hip/hip_runtime.h"
#include <iostream>
#include "dp_model.h"

__device__ void index2xwu(int i, int *dims)
{
    // from index, get x, w, u
    // dim[0, 1, 2] is dim[x, w, u]
    // i = x_idx * (Nw*Nu) + w_idx * (Nu) + u_idx
    // try to avoid using "%"

    // int x_idx = i/(dims[1]*dim[2]);
    // int i_ = i-x_idx*dim[1]*dim[2];
    // int w_idx = i_/dims[2];
    // int u_idx = (i_ - w_idx*dims[2]);
}

// TODO: modify this kernel
// Kernel function to calculate the control/action cost
__global__ void bi_q_kernel(int k, float *x, float *w, float *u, int *t, float *p, float *v, int *a)
{
  //max number of thread possible, some will not be used
  __shared__ float cache[1024];

  // <x, w> -u-> <x2, w2>
  // grid: 3D, <x,w,u>
  //  gridDim.x: N_x;    gridDim.y: N_w;    gridDim.z: N_u
  // blockIdx.x: x_idx; blockIdx.y: w_idx; blockIdx.z: u_idx
  // block: 1D <w>, dimension is not N_w
  //  blockDim.x: 2^m, greater than N_w
  // threadIdx.x: w2_idx
  int x_idx = blockIdx.x;
  int w_idx = blockIdx.y;
  int u_idx = blockIdx.z;
  int n_x = gridDim.x;
  int n_w = gridDim.y;
  int n_u = gridDim.z;
  int w2_idx = threadIdx.x;

  // STEP 1: find the following x_ by given k, x, w, u and the model
  // prepare transition matrix <x_idx, w_idx> -u_idx-> x'_idx
  // input k, x_idx, w_idx, u_idx, output x'_idx
  int x2_idx = t[x_idx*(n_w*n_u)+w_idx*n_u+n_u];

  // STEP 2: by given transition probability matrix, calculate the p*v
  // find p(w -> w') 
  int p_idx = w_idx * n_w + w2_idx;
  // find v(<x',w'>)
  int v_idx = k*(n_x*n_w) + x2_idx * n_w + w2_idx;
  float pv = p[p_idx]*v[v_idx];

  // STEP 3: do the sum reduction here

  // STEP 4: calculate l(k,x,u)
  float l = x[x_idx]*x[x_idx] + u[u_idx]*u[u_idx];

}

// TODO: modify this kernel
// Kernel function to find the control/action with the lowest cost (q-value)
__global__ void bi_min_kernel(int len, int *dims, int k, float *x, float *w, float *u, int *t, float *p, float *v, int *a)
{

}

// Kernel function to calculate the final cost/value at the last step
__global__ void bi_terminal_kernel(int k, float *x, float *w, float *u, int *t, float *p, float *v, int *a)
{
  // <x, w> -u-> <x2, w2>
  // grid: 3D, <x,w,u>
  //  gridDim.x: N_x;    gridDim.y: N_w;    gridDim.z: N_u
  // blockIdx.x: x_idx; blockIdx.y: w_idx; blockIdx.z: u_idx
  // block: 1D <w>, dimension is not N_w
  //  blockDim.x: 2^m, greater than N_w
  // threadIdx.x: w2_idx

  int x_idx = blockIdx.x;
  int w_idx = blockIdx.y;
  int u_idx = blockIdx.z;
  int n_x = gridDim.x;
  int n_w = gridDim.y;
  int n_u = gridDim.z;

  int v_idx = k*(n_x*n_w) + x_idx * n_w + w_idx;
  v[v_idx] = (1-x[x_idx])*(1-x[x_idx]);
}

int gpu_main(DPModel * model)
{
  int n_x = model->x_set.count;
  int n_w = model->w_set.count;
  int n_u = model->u_set.count;
  int N = model->N;

  int len = n_x * n_w * n_u;
  float *x, *w, *u;
  int *t;
  float *p, *v;
  int *a;

  // Allocate Unified Memory . accessible from CPU or GPU
  hipMallocManaged(&x, n_x*sizeof(float));
  hipMallocManaged(&w, n_w*sizeof(float));
  hipMallocManaged(&u, n_u*sizeof(float));
  
  hipMallocManaged(&t, len*sizeof(int));
  // transition probability matrix size: Nw*Nw
  hipMallocManaged(&p, n_w*n_w*sizeof(float));
  // You can do (N+1) for the whole value table, or 2 as a ping-pong buffer
  // The whole value table size will be (N+1)*N_x*N_w
  // Ping-pong buffer type will be 2*N_x*N_w
  hipMallocManaged(&v, (N+1)*n_x*n_w*sizeof(float));
  // You may only need 
  hipMallocManaged(&a, n_x*sizeof(int));

  

  // initialize x, w, and u value to GPU for reference arrays on the host
  memcpy(x, model->x_set.list, n_x*sizeof(float));
  memcpy(w, model->w_set.list, n_w*sizeof(float));
  memcpy(u, model->u_set.list, n_u*sizeof(float));

  // Initialize "index" transition matrix <x,w> -u-> x'
  memcpy(t, model->s_trans_table, n_x*n_w*n_u*sizeof(int));
  // Initialize transition probability matrix w -> w'
  //memcpy(p, model->prob_table, n_x*n_w**sizeof(float));
  for (int i = 0; i < n_w; ++i)
  {
    for (int j=0; j < n_w; ++j)
      p[i*n_w + j] = (i==j)?1:0;
  }

  // Set up parameters for parallel computing
  int blockSize = 1024;
  dim3 grid(n_x, n_w, n_u);

  // Here k = N, the last step
  bi_terminal_kernel<<<grid, blockSize>>>(N, x, w, u, t, p, v, a);
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  for (int k = N-1; k >= 0; k--)
  {
    bi_q_kernel<<<grid, blockSize>>>(k, x, w, u, t, p, v, a);
    hipDeviceSynchronize();
  }

  // Free memory
  hipFree(x);
  hipFree(w);
  hipFree(u);
  hipFree(t);
  hipFree(p);
  hipFree(v);
  hipFree(a);

  return 0;
}